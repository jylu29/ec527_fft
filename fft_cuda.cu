#include "hip/hip_runtime.h"
// nvcc -arch compute_35 fft_cuda.cu -o fft_cuda
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_complex.h>
#include "cuPrintf.cu"

#define MINVAL   0.0
#define MAXVAL  1000.0
#define N 4194304
#define BLOCK_SIZE 1024

/**
 * perform fft_kernel using Cooley–Tukey algorithm
 */
__global__ void fft_kernel(hipComplex *result, int m) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    // Compute the FFT using butterfly operations
    if ((j&m)==0) {
        hipComplex t;
        float tmp = -1.0 * M_PI * j / (1.0 * m);
        t.x = cosf(tmp);
        t.y = sinf(tmp);
        t = hipCmulf(result[j + m], t);
        hipComplex u = result[j];
//    cuPrintf("t = %f + %fi, m=%d\n", t.x, t.y, m);

        result[j] = hipCaddf(u, t);
        result[j + m] = hipCsubf(u, t);
    }
}


double fRand(double fMin, double fMax) {
    double f = (double) random() / RAND_MAX;
    return fMin + f * (fMax - fMin);
}

void initialize(hipComplex *ptr) {
    srandom(N);
    for (int i = 0; i < N; i++) {
        hipComplex c;
        c.y = (float) fRand((double) (MINVAL), (double) (MAXVAL));
        c.x = (float) fRand((double) (MINVAL), (double) (MAXVAL));
//        c.x = i+1;
//        c.y=0;
        ptr[i] = c;
    }
}

double interval(struct timespec start, struct timespec end) {
    struct timespec temp;
    temp.tv_sec = end.tv_sec - start.tv_sec;
    temp.tv_nsec = end.tv_nsec - start.tv_nsec;
    if (temp.tv_nsec < 0) {
        temp.tv_sec = temp.tv_sec - 1;
        temp.tv_nsec = temp.tv_nsec + 1000000000;
    }
    return (((double) temp.tv_sec) + ((double) temp.tv_nsec) * 1.0e-9);
}

//bit-reverse order of d, store in r
__global__ void bit_rev_reorder(hipComplex *__restrict__ r, hipComplex *__restrict__ d, int s) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    r[__brev(id) >> (32 - s)] = d[id];
//    cuPrintf("id %d", id);
}

int main() {
    auto *h_x = (hipComplex *) malloc(sizeof(hipComplex) * N);
    auto *h_result = (hipComplex *) malloc(sizeof(hipComplex) * N);
    hipComplex *d_x, *d_result;
    hipMalloc((void **) &d_x, N * sizeof(hipComplex));
    hipMalloc((void **) &d_result, N * sizeof(hipComplex));
    // define input signal
    initialize(h_x);

    struct timespec time_start, time_stop;
    // compute FFT
    clock_gettime(CLOCK_REALTIME, &time_start);
    hipMemcpy(d_x, h_x, N * sizeof(hipComplex), hipMemcpyHostToDevice);
    hipSetDevice(0);
    hipEvent_t start;
    hipEventCreate(&start);
    hipEventRecord(start, 0);
    cudaPrintfInit();
    for (int i = 0; i < 10; i++) {
        int stages = (int) log2(N);
        // bit_reverse with cuda
        bit_rev_reorder<<<N / BLOCK_SIZE, BLOCK_SIZE>>>(d_result, d_x, stages);
        hipDeviceSynchronize();
        for (int s = 0; s < stages; s++) {
            int m = 1 << s;
//            for (int k = 0; k < N; k += m) {
            fft_kernel<<<(int) (N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_result, m);
                hipDeviceSynchronize();
//            }
        }
    }
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
    hipMemcpy(h_result, d_result, N * sizeof(hipComplex), hipMemcpyDeviceToHost);
    clock_gettime(CLOCK_REALTIME, &time_stop);
    double time = interval(time_start, time_stop);


    // print result
    printf("FFT: \n");
    for (int i = 0; i < 8; i++) {
        printf("%f + %fi\n", h_result[i].x, h_result[i].y);
    }
    printf("\n");
    printf("Time = %8.4g s\n", time);

    return 0;
}

